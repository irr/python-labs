
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sayHello() {
    printf("Hello world from the GPU!\n");
}

int main() {
   printf("Hello world from the CPU!\n");

   sayHello<<<1,1>>>();
   hipDeviceSynchronize();
   
   return 0;
}
